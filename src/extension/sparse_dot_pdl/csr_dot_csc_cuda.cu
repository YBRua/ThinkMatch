#include "hip/hip_runtime.h"
#include "paddle/extension.h"


template <typename scalar_t>
__global__ void csr_dot_csc_cuda_kernel(
    const int64_t* __restrict__ t1_indices,
    const int64_t* __restrict__ t1_indptr,
    const scalar_t* __restrict__ t1_data,
    const int64_t* __restrict__ t2_indices,
    const int64_t* __restrict__ t2_indptr,
    const scalar_t* __restrict__ t2_data,
    scalar_t* __restrict__ out_dense,
    const int64_t out_h,
    const int64_t out_w
)
{
    const int64_t ij = blockIdx.x * blockDim.x + threadIdx.x;
    const int64_t b = blockIdx.y;

    if (ij < out_h * out_w)
    {
        const int64_t i = ij / out_w;
        const int64_t j = ij % out_w;

        const int64_t t1_start = t1_indptr[b * out_h + i];
        const int64_t t1_stop = t1_indptr[b * out_h + i + 1];

        const int64_t t2_start = t2_indptr[b * out_w + j];
        const int64_t t2_stop = t2_indptr[b * out_w + j + 1];

        scalar_t outp = 0;
        int64_t t1_ptr_idx = t1_start;
        int64_t t2_ptr_idx = t2_start;

        while (t1_ptr_idx < t1_stop && t2_ptr_idx < t2_stop)
        {
            int64_t t1_cur_indice = t1_indices[t1_ptr_idx];
            int64_t t2_cur_indice = t2_indices[t2_ptr_idx];
            if (t1_cur_indice == t2_cur_indice)
            {
                outp += t1_data[t1_ptr_idx] * t2_data[t2_ptr_idx];
                t1_ptr_idx++;
                t2_ptr_idx++;
            }
            else if (t1_cur_indice < t2_cur_indice)
                t1_ptr_idx++;
            else
                t2_ptr_idx++;
        }
        out_dense[b * out_w * out_h + i * out_w + j] = outp;
    }
}


std::vector<paddle::Tensor> csr_dot_csc_cuda(
    const paddle::Tensor &t1_indices,
    const paddle::Tensor &t1_indptr,
    const paddle::Tensor &t1_data,
    const paddle::Tensor &t2_indices,
    const paddle::Tensor &t2_indptr,
    const paddle::Tensor &t2_data,
    int64_t batch_size,
    int64_t out_h,
    int64_t out_w
){
    std::vector<int64_t> shape = {batch_size, out_h, out_w};
    auto out_dense = paddle::Tensor(t1_indices.place(), shape);

    const int block = 1024;
    const dim3 grid((out_h * out_w + block - 1) / block, batch_size);

    PD_DISPATCH_FLOATING_AND_HALF_TYPES(t1_data.type(), "csr_dot_csc_cuda", ([&] {
    csr_dot_csc_cuda_kernel<scalar_t><<<grid, block>>>(
        t1_indices.data<int64_t>(),
        t1_indptr.data<int64_t>(),
        t1_data.data<scalar_t>(),
        t2_indices.data<int64_t>(),
        t2_indptr.data<int64_t>(),
        t2_data.data<scalar_t>(),
        out_dense.mutable_data<scalar_t>(),
        out_h,
        out_w);
    }));
    return {out_dense};
}
